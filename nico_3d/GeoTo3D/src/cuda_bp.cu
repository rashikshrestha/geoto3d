#include "hip/hip_runtime.h"
#include <stdio.h>

#include "ncgl.h"

#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
//#include <cutil_inline.h>
#include "cuda_bp.h"

surface<void, 2> outputSurfRef;

int getIndex(int z, int x, int y , int w, int h, int d)
{
    return (y*w+x)*d+z;
}

extern "C" void
normalizeCPU(float* dst , float* src , int w, int h, int d)
{

    //!normalize
    for (unsigned int y=0; y<h; ++y)
        for (unsigned int x=0; x<w; ++x)
        {
            float sum = 0.0f;
            for (unsigned int z=0; z<d; ++z)
                sum +=src[getIndex(z,x,y,w,h,d)];

            for (unsigned int z=0; z<d; ++z)
                dst[getIndex(z,x,y,w,h,d)] = src[getIndex(z,x,y,w,h,d)]/sum;
        }
}

extern "C" void
smoothXYCPU(float* dst , float* src , int w, int h, int d)
{

    for (unsigned int y=0; y<h; ++y)
        for (unsigned int x=0; x<w; ++x)
            for (unsigned int z=0; z<d; ++z)
            {
                float   value   = src[getIndex(z,x,y,w,h,d)];
                int     weight  = 1.0f;

                if (x>0)
                {
                    value+=src[getIndex(z,x-1,y,w,h,d)];
                    weight+=1.0f;
                }
                if (x<w-1)
                {
                    value+=src[getIndex(z,x+1,y,w,h,d)];
                    weight+=1.0f;
                }
                if (y>0)
                {
                    value+=src[getIndex(z,x,y-1,w,h,d)];
                    weight+=1.0f;
                }
                if (y<h-1)
                {
                    value+=src[getIndex(z,x,y+1,w,h,d)];
                    weight+=1.0f;
                }
                dst[getIndex(z,x,y,w,h,d)] = value/weight;
            }
}

extern "C" void
smoothZCPU(float* dst , float* src , int w, int h, int d)
{

    //!smoothZ
    for (unsigned int y=0; y<h; ++y)
        for (unsigned int x=0; x<w; ++x)
            for (unsigned int z=0; z<d; ++z)
            {
                float   value   = src[getIndex(z,x,y,w,h,d)];
                int     weight  = 1.0f;

                if (z>0)
                {
                    value+=src[getIndex(z-1,x,y,w,h,d)];
                    weight+=1.0f;
                }
                if (z<d-1)
                {
                    value+=src[getIndex(z+1,x,y,w,h,d)];
                    weight+=1.0f;
                }

                dst[getIndex(z,x,y,w,h,d)] = value/weight;
            }
}

extern "C" void
compareCPU(float* dst , float* src , int w, int h, int d)
{
    for (int z = 0; z<d; ++z)
        for (int y = 0; y<h; ++y)
            for (int x = 0; x<w; ++x)
                if (fabs(dst[(y*w+x)*d+z]-src[(y*w+x)*d+z])>0.000001)
                    fprintf(stderr,"error at (%d,%d,%d): %f %f\n",x,y,z,dst[(y*w+x)*d+z],src[(y*w+x)*d+z]);
}

__global__ void normalizeKernel(float* src_dst , float* src , int w, int h, int d, float orig_weight)
{

    int x 		= blockIdx.x;
    int y 		= blockIdx.y;
	int tid		= threadIdx.x;
	int offset 	= (y*w+x)*d;

    __shared__ float idata[1024];
    __shared__ float odata[1024];

	float value = src_dst[offset+tid];
	odata[tid] 	= idata[tid] = value*value;
	__syncthreads();

	int cnt = d;
	int hlf = (cnt+1)>>1;

	while (cnt>1) {
		if ((tid<hlf) && (hlf+tid<cnt))
			odata[tid] += odata[hlf+tid];
		__syncthreads();
		cnt = hlf;
		hlf = (cnt+1)>>1;
	}

	float tmp = d*idata[tid]/odata[0];
    if (src)
		tmp = (1.0f-orig_weight)*tmp + orig_weight*src[offset+tid]*src[offset+tid];
	src_dst[offset+tid] = tmp;

}

__global__ void smoothXYKernel(float* dst , float* src , int w, int h, int d, int range)
{

    int x 		= blockIdx.x;
    int y 		= blockIdx.y;
	int tid		= threadIdx.x;
	int offset 	= (y*w+x)*d;

    __shared__ float odata[1024];

    odata[tid] = 0.0f;

    float   tmp = 3.14159265359f/(range+1);
    float 	weight;
    float 	weight_total = 0.0f;

    int ox_min = -min(range , x		);
    int ox_max = +min(range , w-1-x	);
    int oy_min = -min(range , y		);
    int oy_max = +min(range , h-1-y	);

    for (int oy=oy_min; oy<=oy_max; ++oy)
        for (int ox=ox_min; ox<=ox_max; ++ox)
        {
	        weight		 = (0.5f + 0.5f*cos(tmp*ox))*(0.5f + 0.5f*cos(tmp*oy));
            odata[tid]	+= weight*src[((y+oy)*w+(x+ox))*d+tid];
            weight_total+= weight;
		}
	dst[offset+tid] = odata[tid]/weight_total;

}

#if 1

__global__ void smoothZKernel(float* dst , float* src , int w, int h, int d , int range)
{

    int x 		= blockIdx.x;
    int y 		= blockIdx.y;
    int tid		= threadIdx.x;
	int offset 	= (y*w+x)*d;

    float   tmp = 3.14159265359f/(range+1);
    float 	weight;
    //float 	total_weight = 0.0f;

    __shared__ float idata[1024];
    __shared__ float odata[1024];

    idata[tid] = src[offset + tid];
    odata[tid] = src[offset + tid];
    __syncthreads();

    for (int r=1; r<=range; ++r)
    {
	    weight		= 0.5f + 0.5f*cos(tmp*r);
       	odata[tid]  = max(odata[tid],weight*idata[max(tid-r,0  )]);
	    __syncthreads();
       	odata[tid]  = max(odata[tid],weight*idata[min(tid+r,d-1)]);
    	__syncthreads();
	}

	dst[offset + tid] = odata[tid];

}

#else

__global__ void smoothZKernel(float* dst , float* src , int w, int h, int d , int range)
{

    int x 		= blockIdx.x;
    int y 		= blockIdx.y;
    int tid		= threadIdx.x;
	int offset 	= (y*w+x)*d;

    float   tmp = 3.14159265359f/(range+1);
    float 	weight;
    float 	total_weight = 1.0f;

    __shared__ float idata[1024];
    __shared__ float odata[1024];

    idata[tid] = src[offset + tid];
    odata[tid] = src[offset + tid];
    __syncthreads();

    for (int r=1; r<=range; ++r)
    {
	    weight		 = 0.5f + 0.5f*cos(tmp*r);
       	odata[tid] 	+= weight*idata[max(tid-r,0  )];
		total_weight+= weight;
	    __syncthreads();
       	odata[tid] 	+= weight*idata[min(tid+r,d-1)];
		total_weight+= weight;
    	__syncthreads();
	}
   	odata[tid] /= total_weight;
   	__syncthreads();

	dst[offset + tid] = odata[tid];

}

#endif

__global__ void extractKernel(float* src , int w, int h, int d, float* zlut_dev)
{
    int x 		= blockIdx.x;
    int y 		= blockIdx.y;
	int tid		= threadIdx.x;
	int offset 	= (y*w+x)*d;

    __shared__ int 		idata[1024];
    __shared__ float 	fdata[1024];

	idata[tid] 	= tid;
	fdata[tid]	= src[offset+tid];
	__syncthreads();

	int cnt = d;
	int hlf = (cnt+1)>>1;

	while (cnt>1) {
		if ((tid<hlf) && (hlf+tid<cnt))
			if (fdata[tid]<fdata[hlf+tid])
			{
				fdata[tid] = fdata[hlf+tid];
				idata[tid] = idata[hlf+tid];
			}
		__syncthreads();
		cnt = hlf;
		hlf = (cnt+1)>>1;
	}

	if (tid == 0)
    {
#if 0
            //! interpolate
            // http://stackoverflow.com/questions/717762/how-to-calculate-the-vertex-of-a-parabola-given-three-points

            float x1 = bdepth[tid]-1;
            float x2 = bdepth[tid]  ;
            float x3 = bdepth[tid]+1;

            float y1 = src[(y*w+x)*d + max(  0, bdepth[tid]-1) ];
            float y2 = src[(y*w+x)*d +          bdepth[tid]    ];
            float y3 = src[(y*w+x)*d + min(d-1, bdepth[tid]+1) ];

            float center;
            if ((y1==y2) && (y1==y3))
                center = x2;
            else
            {
                float B         = (x3*x3 * (y1 - y2) + x2*x2 * (y3 - y1) + x1*x1 * (y2 - y3));
                float A         = (x3 * (y2 - y1) + x2 * (y1 - y3) + x1 * (y3 - y2));
                center          = -0.5f*B/A;
            }

            float2 fColor  = make_float2((center+0.5f)/d , 1.0f);
            surf2Dwrite(fColor, outputSurfRef, x * sizeof(float2), y);
#else

	float resdepth = zlut_dev[idata[0]];
	float2 fColor  = make_float2(resdepth , 1.0f);
    surf2Dwrite(fColor, outputSurfRef, x * sizeof(float2), y);
#endif
	}
}

int my_it = 0;

extern "C" void
save(float* src , int w, int h, int d) {

	if (my_it == 5) {

		int 	nrpix2d = w*h;
		int 	nrpix3d = nrpix2d*d;
		int 	nrdat3d	= nrpix3d*sizeof(float);
		float* 	tmp 	= (float*)malloc(nrdat3d);

		hipMemcpy(tmp,src,nrdat3d,hipMemcpyDeviceToHost);

		float 	maxval 	= 0.0f;
		for (int i=0;i<nrpix3d;++i)
			maxval = std::max(maxval,tmp[i]);

		fprintf(stderr,"maxval = %f\n",maxval);

		float fact = 255.0f;///maxval;

		ncImageBuffer<unsigned char> imbuf(w,h);
		for (int z=0;z<d;++z)
		{
			float* slice = &tmp[z];
			for (int p=0;p<nrpix2d;++p)
			{
				imbuf[p] = (int)(fact*(*slice));
				slice+=d;
			}

			char imname[1024];
			sprintf(imname,"/tmp/%04d_%04d.png",my_it,z);
			ncc::writeImage(imname,imbuf);
		}

		free(tmp);
	}

    my_it++;
}

extern "C" void
beliefPropagation(hipArray* my_array, float* idata0 , float* idata1 , float* idata2 , int w, int h, int d , float* zlut_dev)
{

    int nrit        = 4;
    int smoothZ     = 32;
    int smoothXY    = 1;

    float init_orig_weight = 0.0f;
    float exit_orig_weight = 0.25f;

    cudaBindSurfaceToArray(outputSurfRef, my_array);
	//save(idata0 , w, h, d);
    normalizeKernel<<<dim3(w,h,1) , dim3(d,1,1)>>>(idata0 , 0 , w, h, d, 0.0f);
	hipMemcpy(idata1,idata0,w*h*d*sizeof(float),hipMemcpyDeviceToDevice);

    for (unsigned int it=0; it<nrit; ++it)
    {
        smoothZKernel 	<<<dim3(w,h,1) , dim3(d,1,1)>>>(idata2 , idata0 , w, h, d, smoothZ);
        smoothXYKernel	<<<dim3(w,h,1) , dim3(d,1,1)>>>(idata0 , idata2 , w, h, d, smoothXY);

        float lrp = (float(it)/float(nrit-1));
        normalizeKernel	<<<dim3(w,h,1) , dim3(d,1,1)>>>(idata0 , idata1 , w, h, d, (1.0f-lrp)*init_orig_weight + lrp*exit_orig_weight);
    }

    extractKernel<<<dim3(w,h,1) , dim3(d,1,1)>>>(idata0 , w, h, d, zlut_dev);
    //cutilCheckMsg("kernel launch failure");
}

