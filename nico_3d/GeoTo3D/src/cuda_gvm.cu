#include "hip/hip_runtime.h"
#include <stdio.h>

#include "ncgl.h"

#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
//#include <cutil_inline.h>
#include "cuda_gvm.h"

#define Xsize 16
#define Ysize 16

surface<void, 2> outputSurfRef;

__shared__ cudaGVM  gvm;
__shared__ int      nrsteps;
__shared__ float3   campos;
__shared__ int3     indexx[Ysize][Xsize];

struct privar {
    uint    vindexx;
    float3  dir3D;
    int		level;
    bool 	inside;
    float3 	f1;
    float3 	f2;
    float 	t;
    float3  bbox_p1;
    float3  bbox_sz;
    float3 	bbox_step;
};


inline  __device__ int3 truncf(float3 a) {
    return make_int3(truncf(a.x),truncf(a.y),truncf(a.z));
}

inline  __device__ int3 operator+(int3 a, int b) {
    return make_int3(a.x+b,a.y+b,a.z+b);
}

inline  __device__ float3 operator+(float3 a, float3 b) {
    return make_float3(a.x+b.x,a.y+b.y,a.z+b.z);
}

inline  __device__ float3 operator-(float3 a, float3 b) {
    return make_float3(a.x-b.x,a.y-b.y,a.z-b.z);
}

inline  __device__ float3 operator-(int3 a, float3 b) {
    return make_float3(a.x-b.x,a.y-b.y,a.z-b.z);
}

inline  __device__ float3 operator*(float a, float3 b) {
    return make_float3(a*b.x,a*b.y,a*b.z);
}

inline  __device__ float3 operator/(float3 a, float3 b) {
    return make_float3(a.x/b.x , a.y/b.y , a.z/b.z);
}

inline  __device__ float3 operator/(float3 a, int b) {
    return make_float3(a.x/b , a.y/b , a.z/b);
}

inline  __device__ int3 max(int3 a, int3 b) {
    return make_int3(max(a.x,b.x) , max(a.y,b.y) , max(a.z,b.z));
}

inline  __device__ int3 floor(float3 a) {
    return make_int3(floor(a.x) , floor(a.y) , floor(a.z));
}


int3 __device__ realIndex(privar & pdata) {
    int3 i_out;
    i_out.x = (pdata.dir3D.x<0.0f ? nrsteps-1-indexx[threadIdx.y][threadIdx.x].x : indexx[threadIdx.y][threadIdx.x].x);
    i_out.y = (pdata.dir3D.y<0.0f ? nrsteps-1-indexx[threadIdx.y][threadIdx.x].y : indexx[threadIdx.y][threadIdx.x].y);
    i_out.z = (pdata.dir3D.z<0.0f ? nrsteps-1-indexx[threadIdx.y][threadIdx.x].z : indexx[threadIdx.y][threadIdx.x].z);
    return i_out;
}

bool __device__ testIndex(privar & pdata) {
	return ((indexx[threadIdx.y][threadIdx.x].x>=0) && (indexx[threadIdx.y][threadIdx.x].x<nrsteps) && (indexx[threadIdx.y][threadIdx.x].y>=0) && (indexx[threadIdx.y][threadIdx.x].y<nrsteps) && (indexx[threadIdx.y][threadIdx.x].z>=0) && (indexx[threadIdx.y][threadIdx.x].z<nrsteps));
}

bool __device__ testVox(privar & pdata) {
	if (pdata.inside)
	{
	    int3 tmp = realIndex(pdata);

		pdata.vindexx   = uint(0);
		pdata.level 	  = gvm.maxlevel;
		while ((gvm.voxels[pdata.vindexx].child>0) && (pdata.level >(gvm.maxlevel-gvm.lod)))
		{
			pdata.level--;
			int offset = 0;
			offset|= ((tmp.x>>pdata.level)&0x1)<<0;
			offset|= ((tmp.y>>pdata.level)&0x1)<<1;
			offset|= ((tmp.z>>pdata.level)&0x1)<<2;
			//offset = bitfieldInsert(offset, (tmp.x>>pdata.level), 0 , 1);
			//offset = bitfieldInsert(offset, (tmp.y>>pdata.level), 1 , 1);
			//offset = bitfieldInsert(offset, (tmp.z>>pdata.level), 2 , 1);
			pdata.vindexx = gvm.voxels[pdata.vindexx].child+offset;
		}

		int mask = ~((1<<pdata.level)-1);
		indexx[threadIdx.y][threadIdx.x].x &= mask;
		indexx[threadIdx.y][threadIdx.x].y &= mask;
		indexx[threadIdx.y][threadIdx.x].z &= mask;

		return ((gvm.voxels[pdata.vindexx].carve&0xFF) == 0x00);
	}
	else
		return true;
};

void __device__ outerIntersect(privar & pdata) {

	float3  tmp = -1.0f*(pdata.f1/pdata.f2);
	pdata.t           = max(tmp.x,max(tmp.y,tmp.z));
	indexx[threadIdx.y][threadIdx.x] 	    = (int3)truncf(pdata.f1+pdata.t*pdata.f2);
}

void __device__ innerIntersect(privar & pdata) {

	int3   planes  = indexx[threadIdx.y][threadIdx.x]+(1<<pdata.level);
	float3    tmp  = (planes-pdata.f1)/pdata.f2;

            pdata.t       = min(tmp.x,min(tmp.y,tmp.z));
	int3   result 	= (int3)floor(pdata.f1+pdata.t*pdata.f2);

	if (pdata.t==tmp.x)
	{
		result.x= planes.x;
		pdata.inside  = result.x<nrsteps;
	}
	else if (pdata.t==tmp.y)
	{
		result.y= planes.y;
		pdata.inside  = result.y<nrsteps;
	}
	else
	{
		result.z= planes.z;
		pdata.inside  = result.z<nrsteps;
	}

	indexx[threadIdx.y][threadIdx.x] = max(indexx[threadIdx.y][threadIdx.x],result);

}

void __device__ processPixel(float3 & p3 , bool & valid , privar & pdata) {

    pdata.bbox_p1     = gvm.bpos;
    pdata.bbox_sz     = gvm.bsize;

    if (pdata.dir3D.x<0.0f)
    {
        pdata.bbox_p1.x+=  pdata.bbox_sz.x;
        pdata.bbox_sz.x = -pdata.bbox_sz.x;
    }
    if (pdata.dir3D.y<0.0f)
    {
        pdata.bbox_p1.y+=  pdata.bbox_sz.y;
        pdata.bbox_sz.y = -pdata.bbox_sz.y;
    }
    if (pdata.dir3D.z<0.0f)
    {
        pdata.bbox_p1.z+=  pdata.bbox_sz.z;
        pdata.bbox_sz.z = -pdata.bbox_sz.z;
    }

	pdata.bbox_step 	= pdata.bbox_sz/nrsteps;

	pdata.f1 			= (campos-pdata.bbox_p1)/pdata.bbox_step;
	pdata.f2			= pdata.dir3D/pdata.bbox_step;

	indexx[threadIdx.y][threadIdx.x] 	        = (int3)floor(pdata.f1);
	bool    test 	= true;
	valid 	        = false;

	pdata.inside = testIndex(pdata);
	if (!pdata.inside)
	{
		outerIntersect(pdata);
		pdata.inside 	= (pdata.t>0.0f) && testIndex(pdata);
		valid	= (testVox(pdata) != test);
	}
	else
	{
		test = testVox(pdata);
	}

	int it = 0;
	while ((pdata.inside) && (!valid) && (it<1000))
	{
		innerIntersect(pdata);
		valid  = (!testVox(pdata));// != test);
		it++;
	}

    p3 = campos+pdata.t*pdata.dir3D;
}





__global__ void drawGVM(unsigned int w, unsigned int h , cudaGVM gvm_) {
    privar pdata;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if ((threadIdx.x==0) && (threadIdx.y==0))
    {
        gvm         = gvm_;
        nrsteps		= (1<<gvm.maxlevel);
        campos      = make_float3(gvm.MV[3].x,gvm.MV[3].y,gvm.MV[3].z);
    }
    __syncthreads();

    if ((x<w) && (y<h))
    {
        uchar4 fColor   = make_uchar4(0,0,0,0);
        float4 tmp      = gvm.Ki*make_float4((2.0f*x+1.0f)/w - 1.0f,(2.0f*y+1.0f)/h - 1.0f, -1.0f, 1.0f);
        tmp.x /= tmp.z;
        tmp.y /= tmp.z;
        tmp.z  = 1.0f;
        tmp.w  = 0.0f;
        tmp    = gvm.MV*tmp;
        pdata.dir3D  = make_float3(tmp.x,tmp.y,tmp.z);

        float3 p3;
        bool valid;
        processPixel(p3,valid,pdata);

        if (valid && (pdata.level==0))
        {
            int tmp = gvm.voxels[pdata.vindexx].carve;
            fColor  = make_uchar4((tmp>>24)&0xFF, (tmp>>16)&0xFF , (tmp>>8)&0xFF , 255);
            //float4 p2D = MVP*p4;
            //p2D/=p2D.w;
            //gl_FragDepth = 0.5f+0.5f*p2D.z;
        }

        surf2Dwrite(fColor, outputSurfRef, x * sizeof(uchar4), y);
    }
}

extern "C" void
runTest(hipArray* my_array, unsigned int w, unsigned int h , cudaGVM gvm) {
    cudaBindSurfaceToArray(outputSurfRef, my_array);
    drawGVM<<<dim3((w+Xsize-1)/Xsize,(h+Ysize-1)/Ysize,1), dim3(Xsize,Ysize,1)>>>(w,h,gvm);
    //cutilCheckMsg("kernel launch failure");
}

